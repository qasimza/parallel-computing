#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"
//#include "utils.h"

const int N= 1024;	// matrix size will be NxN
const int K= 16;	// threadBlock size KxK
int compare_matrices(float *gpu, float *ref, int N)
{
        int result = 0;
        for(int j=0; j < N; j++)
        for(int i=0; i < N; i++)
                if (ref[i + j*N] != gpu[i + j*N])
                   {result = 1;}
 return result;
}


// fill a matrix with sequential numbers in the range 0..N-1
void fill_matrix(float *mat, int N)
{
        for(int j=0; j < N * N; j++)
                mat[j] = (float) j;
}

// The following functions and kernels are for your references
void 
transpose_CPU(float in[], float out[])
{
	for(int j=0; j < N; j++)
    	for(int i=0; i < N; i++)
      		out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

// to be launched on a single thread
__global__ void 
transpose_serial(float in[], float out[])
{
	for(int j=0; j < N; j++)
		for(int i=0; i < N; i++)
			out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

// to be launched with one thread per row of output matrix
__global__ void 
transpose_parallel_per_row(float in[], float out[])
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	for(int j=0; j < N; j++)
		out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}



// Write two tiled versions of transpose -- One using shared memory. 
// To be launched with one thread per element, in KxK threadblocks.
// You will determine for each thread (x,y) in tile the element (i,j) of global output matrix. 

__global__ void 
transpose_parallel_per_element_tiled(float in[], float out[])
{
	int i = threadIdx.x + blockIdx.x * K;
	int j = threadIdx.y + blockIdx.y * K;
	out[j+i*N] = in[i+j*N];
}

__global__ void 
transpose_parallel_per_element_tiled_shared(float in[], float out[])
{
	__shared__ float tile[K][K];
	
	int in_i = K * blockIdx.x;
	int out_i = K * blockIdx.y;
	int in_j = K * blockIdx.y;
	int out_j = K * blockIdx.x;
	int x = threadIdx.x;
	int y = threadIdx.y;

	tile[y][x] = in[in_i +x + N * (in_j + y)];
	__syncthreads();
	out[out_i + x + N * (out_j + y)] = tile[x][y];	
}

int main(int argc, char **argv)
{
	int numbytes = N * N * sizeof(float);

	float *in = (float *) malloc(numbytes);
	float *out = (float *) malloc(numbytes);
	float *gold = (float *) malloc(numbytes);

	fill_matrix(in, N);
	transpose_CPU(in, gold);

	float *d_in, *d_out;

	hipMalloc(&d_in, numbytes);
	hipMalloc(&d_out, numbytes);
	hipMemcpy(d_in, in, numbytes, hipMemcpyHostToDevice);

	GpuTimer timer;


    timer.Start();
	transpose_serial<<<1,1>>>(d_in, d_out);
	timer.Stop();
    for (int i=0; i < N*N; ++i){out[i] = 0.0;}
    hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_serial: %g ms.\nVerifying ...%s\n", 
		   timer.Elapsed(), compare_matrices(out, gold, N) ? "Failed" : "Success");

   
    hipMemcpy(d_out, d_in, numbytes, hipMemcpyDeviceToDevice); //clean d_out
    timer.Start();
	transpose_parallel_per_row<<<1,N>>>(d_in, d_out);
	timer.Stop();
    for (int i=0; i < N*N; ++i){out[i] = 0.0;}  //clean out
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_row: %g ms.\nVerifying ...%s\n", 
		    timer.Elapsed(), compare_matrices(out, gold, N) ? "Failed" : "Success");

    hipMemcpy(d_out, d_in, numbytes, hipMemcpyDeviceToDevice); //clean d_out
    // Tiled versions
    const int K=16;
    dim3 blocks_tiled(N/K,N/K);
	dim3 threads_tiled(K,K);
	timer.Start();
	transpose_parallel_per_element_tiled<<<blocks_tiled,threads_tiled>>>(d_in, d_out);
	timer.Stop();
    for (int i=0; i < N*N; ++i){out[i] = 0.0;}
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_element_tiled %dx%d: %g ms.\nVerifying ...%s\n", 
		   K, K, timer.Elapsed(), compare_matrices(out, gold, N) ? "Failed" : "Success");

    hipMemcpy(d_out, d_in, numbytes, hipMemcpyDeviceToDevice); //clean d_out
    dim3 blocks_tiled_sh(N/K,N/K);
	dim3 threads_tiled_sh(K,K);
     timer.Start();
	transpose_parallel_per_element_tiled_shared<<<blocks_tiled_sh,threads_tiled_sh>>>(d_in, d_out);
	timer.Stop();
    for (int i=0; i < N*N; ++i){out[i] = 0.0;}
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_element_tiled_shared %dx%d: %g ms.\nVerifying ...%s\n", 
		   K, K, timer.Elapsed(), compare_matrices(out, gold, N) ? "Failed" : "Success");

	hipFree(d_in);
	hipFree(d_out);
}